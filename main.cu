#include "hip/hip_runtime.h"
/*
 * main.cpp
 *
 *  Created on: Apr 17, 2012
 *      Author: Wyatt
 */

#include "MeshLoader.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <iostream>
#include "Image.h"
#include <math.h>
#include <hip/hip_runtime.h>

#define MY_WINDOW_SIZE 2000
#define NUM_THREADS 512

#define round_up(x, y) (((x) % (y) == 0) ? ((x)/(y)) : (((x)/(y)) + 1))
#define max(x1, x2) ((x1) > (x2) ? (x1) : (x2))
#define min(x1, x2) ((x1) < (x2) ? (x1) : (x2))

float redColorBuffer[MY_WINDOW_SIZE * MY_WINDOW_SIZE];
float greenColorBuffer[MY_WINDOW_SIZE * MY_WINDOW_SIZE];
float blueColorBuffer[MY_WINDOW_SIZE * MY_WINDOW_SIZE];

int depthBuffer[MY_WINDOW_SIZE * MY_WINDOW_SIZE];
//
//__device__ glm::ivec2 worldToScreen(glm::vec2 point)
//{
//   glm::ivec2 newPoint(0);
//
//    point *= 4;
//
//   newPoint.x = ((point.x + 1) * (MY_WINDOW_SIZE - 1)) / 2;
//   newPoint.y = ((point.y + 1) * (MY_WINDOW_SIZE - 1)) / 2;
////    newPoint.y = (MY_WINDOW_SIZE - 1) - ((point.y + 1) * (MY_WINDOW_SIZE - 1)) / 2;
//
//   return newPoint;
//}
//
//__device__ ScreenTriangle triangleToScreenSpace(Triangle tri)
//{
//   ScreenTriangle screenTri;
//
//   screenTri.v0.x = ((tri.v0.x + 1) * (MY_WINDOW_SIZE - 1)) / 2;
//   screenTri.v0.y = ((tri.v0.y + 1) * (MY_WINDOW_SIZE - 1)) / 2;
//   screenTri.v0z = tri.v0.z;
//
//   screenTri.v1.x = ((tri.v1.x + 1) * (MY_WINDOW_SIZE - 1)) / 2;
//   screenTri.v1.y = ((tri.v1.y + 1) * (MY_WINDOW_SIZE - 1)) / 2;
//    screenTri.v1z = tri.v1.z;
//
//   screenTri.v2.x = ((tri.v2.x + 1) * (MY_WINDOW_SIZE - 1)) / 2;
//   screenTri.v2.y = ((tri.v2.y + 1) * (MY_WINDOW_SIZE - 1)) / 2;
//    screenTri.v2z = tri.v2.z;
//
//    screenTri.normal = glm::normalize(glm::cross(glm::normalize(tri.v1 - tri.v0), glm::normalize(tri.v2 - tri.v0)));
//
//   screenTri.topLeft.x = min(min(screenTri.v0.x, screenTri.v1.x), screenTri.v2.x);
//    screenTri.topLeft.y = min(min(screenTri.v0.y, screenTri.v1.y), screenTri.v2.y);
//
//    screenTri.bottomRight.x = max(max(screenTri.v0.x, screenTri.v1.x), screenTri.v2.x);
//    screenTri.bottomRight.y = max(max(screenTri.v0.y, screenTri.v1.y), screenTri.v2.y);
//
//   return screenTri;
//}

__global__ void drawTriangle(Triangle *tris, int size, float *redColorBuffer_d, float *greenColorBuffer_d,
      float *blueColorBuffer_d, int *depthBuffer_d)
{
   if (blockIdx.x * blockDim.x + threadIdx.x >= size) {
      return;
   }

   Triangle tri = tris[blockIdx.x * blockDim.x + threadIdx.x];
   /*
   tri.v0 = glm::vec3(0.0, 0.5, 0.0);
   tri.v1 = glm::vec3(-0.5, -0.5, 0.0);
   tri.v2 = glm::vec3(0.5, -0.25, 0.0);
   */

   ScreenTriangle screenTri;

   screenTri.v0.x = ((tri.v0.x + 1) * (MY_WINDOW_SIZE - 1)) / 2;
   screenTri.v0.y = ((tri.v0.y + 1) * (MY_WINDOW_SIZE - 1)) / 2;
   screenTri.v0z = tri.v0.z;

   screenTri.v1.x = ((tri.v1.x + 1) * (MY_WINDOW_SIZE - 1)) / 2;
   screenTri.v1.y = ((tri.v1.y + 1) * (MY_WINDOW_SIZE - 1)) / 2;
    screenTri.v1z = tri.v1.z;

   screenTri.v2.x = ((tri.v2.x + 1) * (MY_WINDOW_SIZE - 1)) / 2;
   screenTri.v2.y = ((tri.v2.y + 1) * (MY_WINDOW_SIZE - 1)) / 2;
    screenTri.v2z = tri.v2.z;

    screenTri.normal = glm::normalize(glm::cross(glm::normalize(tri.v1 - tri.v0), glm::normalize(tri.v2 - tri.v0)));

   screenTri.topLeft.x = min(min(screenTri.v0.x, screenTri.v1.x), screenTri.v2.x);
    screenTri.topLeft.y = min(min(screenTri.v0.y, screenTri.v1.y), screenTri.v2.y);

    screenTri.bottomRight.x = max(max(screenTri.v0.x, screenTri.v1.x), screenTri.v2.x);
    screenTri.bottomRight.y = max(max(screenTri.v0.y, screenTri.v1.y), screenTri.v2.y);

   glm::vec3 light = glm::normalize(glm::vec3(1.0, 0.0, 1.0));

    float area = glm::determinant(glm::mat2(screenTri.v2 - screenTri.v0, screenTri.v1 - screenTri.v0));
    float color = max(glm::dot(screenTri.normal, light), 0.0f);

    for (int y = screenTri.topLeft.y; y < screenTri.bottomRight.y; ++y)
    {
        for (int x = screenTri.topLeft.x; x < screenTri.bottomRight.x; ++x)
        {
            if (x >= MY_WINDOW_SIZE || y >= MY_WINDOW_SIZE)
            {
                continue;
            }

            glm::ivec2 point(x, y);

            float a = glm::determinant(glm::mat2(screenTri.v2 - point, screenTri.v1 - point)) / area;
            float b = glm::determinant(glm::mat2(screenTri.v1 - point, screenTri.v0 - point)) / area;
            float g = glm::determinant(glm::mat2(screenTri.v0 - point, screenTri.v2 - point)) / area;

            if (a < 0 || b  < 0 || g < 0)
            {
                continue;
            }

            int depth = (a * screenTri.v0z + b * screenTri.v1z + g * screenTri.v2z) * 10000;

            if (depth > atomicMax(&(depthBuffer_d[(y * MY_WINDOW_SIZE) + x]), depth))
            {
                //depthBuffer_d[(y * MY_WINDOW_SIZE) + x] = depth;

                redColorBuffer_d[(y * MY_WINDOW_SIZE) + x] = color;
                greenColorBuffer_d[(y * MY_WINDOW_SIZE) + x] = color;
                blueColorBuffer_d[(y * MY_WINDOW_SIZE) + x] = color;
            }
        }
    }
}

void writeImageToFile()
{
   Image img(MY_WINDOW_SIZE, MY_WINDOW_SIZE);

   for (int y = 0; y < MY_WINDOW_SIZE; ++y)
   {
      for (int x = 0; x < MY_WINDOW_SIZE; ++x)
      {
         color_t col;
         col.r = redColorBuffer[(y * MY_WINDOW_SIZE) + x];
         col.g = greenColorBuffer[(y * MY_WINDOW_SIZE) + x];
         col.b = blueColorBuffer[(y * MY_WINDOW_SIZE) + x];
         col.f = 1.0f;


         img.pixel(x, y, col);
      }
   }

   img.WriteTga("IMG.tga", false);
}

int main()
{
   float *redColorBuffer_d, *greenColorBuffer_d, *blueColorBuffer_d;
   int *depthBuffer_d;
   Triangle *triangleBuffer_d;

   hipMalloc((void **) &redColorBuffer_d, sizeof(float) * MY_WINDOW_SIZE * MY_WINDOW_SIZE);
   hipMalloc((void **) &greenColorBuffer_d, sizeof(float) * MY_WINDOW_SIZE * MY_WINDOW_SIZE);
   hipMalloc((void **) &blueColorBuffer_d, sizeof(float) * MY_WINDOW_SIZE * MY_WINDOW_SIZE);
   hipMalloc((void **) &depthBuffer_d, sizeof(int) * MY_WINDOW_SIZE * MY_WINDOW_SIZE);

    for (int i = 0; i < MY_WINDOW_SIZE * MY_WINDOW_SIZE; ++i)
    {
        redColorBuffer[i] = 0.0;
        greenColorBuffer[i] = 1.0;
        blueColorBuffer[i] = 0.0;
        depthBuffer[i] = -2147483648;
//      depthBuffer[i] = FLT_MIN;
    }

   //memset(redColorBuffer, 0.0, sizeof(float) * MY_WINDOW_SIZE * MY_WINDOW_SIZE);
   hipMemcpy(redColorBuffer_d, redColorBuffer,
    sizeof(float) * MY_WINDOW_SIZE * MY_WINDOW_SIZE, hipMemcpyHostToDevice);

   //memset(greenColorBuffer, 1.0f, sizeof(float) * MY_WINDOW_SIZE * MY_WINDOW_SIZE);
   hipMemcpy(greenColorBuffer_d, greenColorBuffer,
    sizeof(float) * MY_WINDOW_SIZE * MY_WINDOW_SIZE, hipMemcpyHostToDevice);

   //memset(blueColorBuffer, 0.0, sizeof(float) * MY_WINDOW_SIZE * MY_WINDOW_SIZE);
   hipMemcpy(blueColorBuffer_d, blueColorBuffer,
    sizeof(float) * MY_WINDOW_SIZE * MY_WINDOW_SIZE, hipMemcpyHostToDevice);

   //memset(depthBuffer, –214748364, sizeof(int) * MY_WINDOW_SIZE * MY_WINDOW_SIZE);
   hipMemcpy(depthBuffer_d, depthBuffer,
    sizeof(int) * MY_WINDOW_SIZE * MY_WINDOW_SIZE, hipMemcpyHostToDevice);


   Mesh mesh = loadMesh("bunny10k.m");
   hipMalloc((void **) &triangleBuffer_d, sizeof(Triangle) * mesh.tris.size());
   hipMemcpy(triangleBuffer_d, &(mesh.tris[0]), sizeof(Triangle) * mesh.tris.size(), hipMemcpyHostToDevice);

   int numBlocks = round_up(mesh.tris.size(), NUM_THREADS);


//    for (int i = 0; i < mesh.tris.size(); ++i)
   {
      //std::cout << "Triangle: " << i << std::endl;
      drawTriangle<<<numBlocks, NUM_THREADS>>>(triangleBuffer_d, mesh.tris.size(), redColorBuffer_d, greenColorBuffer_d, blueColorBuffer_d, depthBuffer_d);
   }

   hipMemcpy(redColorBuffer, redColorBuffer_d,
    sizeof(float) * MY_WINDOW_SIZE * MY_WINDOW_SIZE, hipMemcpyDeviceToHost);

   hipMemcpy(greenColorBuffer, greenColorBuffer_d,
    sizeof(float) * MY_WINDOW_SIZE * MY_WINDOW_SIZE, hipMemcpyDeviceToHost);

   hipMemcpy(blueColorBuffer, blueColorBuffer_d,
    sizeof(float) * MY_WINDOW_SIZE * MY_WINDOW_SIZE, hipMemcpyDeviceToHost);

   writeImageToFile();
   
   hipFree(redColorBuffer_d);
   hipFree(greenColorBuffer_d);
   hipFree(blueColorBuffer_d);
   hipFree(depthBuffer_d);

   printf("Done!");

   return 0;
}
