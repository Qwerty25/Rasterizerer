#include "hip/hip_runtime.h"
/*
 * main.cpp
 *
 *  Created on: Apr 17, 2012
 *      Author: Wyatt
 */

#include "MeshLoader.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <iostream>
#include "Image.h"
#include <math.h>
#include <hip/hip_runtime.h>

#define MY_WINDOW_SIZE 512

#define max(x1, x2) ((x1) > (x2) ? (x1) : (x2))
#define min(x1, x2) ((x1) < (x2) ? (x1) : (x2))

float redColorBuffer[MY_WINDOW_SIZE * MY_WINDOW_SIZE];
float greenColorBuffer[MY_WINDOW_SIZE * MY_WINDOW_SIZE];
float blueColorBuffer[MY_WINDOW_SIZE * MY_WINDOW_SIZE];

float depthBuffer[MY_WINDOW_SIZE * MY_WINDOW_SIZE];
//
//__device__ glm::ivec2 worldToScreen(glm::vec2 point)
//{
//	glm::ivec2 newPoint(0);
//
//    point *= 4;
//
//	newPoint.x = ((point.x + 1) * (MY_WINDOW_SIZE - 1)) / 2;
//	newPoint.y = ((point.y + 1) * (MY_WINDOW_SIZE - 1)) / 2;
////    newPoint.y = (MY_WINDOW_SIZE - 1) - ((point.y + 1) * (MY_WINDOW_SIZE - 1)) / 2;
//
//	return newPoint;
//}
//
//__device__ ScreenTriangle triangleToScreenSpace(Triangle tri)
//{
//	ScreenTriangle screenTri;
//
//	screenTri.v0.x = ((tri.v0.x + 1) * (MY_WINDOW_SIZE - 1)) / 2;
//	screenTri.v0.y = ((tri.v0.y + 1) * (MY_WINDOW_SIZE - 1)) / 2;
//	screenTri.v0z = tri.v0.z;
//
//	screenTri.v1.x = ((tri.v1.x + 1) * (MY_WINDOW_SIZE - 1)) / 2;
//	screenTri.v1.y = ((tri.v1.y + 1) * (MY_WINDOW_SIZE - 1)) / 2;
//    screenTri.v1z = tri.v1.z;
//
//	screenTri.v2.x = ((tri.v2.x + 1) * (MY_WINDOW_SIZE - 1)) / 2;
//	screenTri.v2.y = ((tri.v2.y + 1) * (MY_WINDOW_SIZE - 1)) / 2;
//    screenTri.v2z = tri.v2.z;
//
//    screenTri.normal = glm::normalize(glm::cross(glm::normalize(tri.v1 - tri.v0), glm::normalize(tri.v2 - tri.v0)));
//
//	screenTri.topLeft.x = min(min(screenTri.v0.x, screenTri.v1.x), screenTri.v2.x);
//    screenTri.topLeft.y = min(min(screenTri.v0.y, screenTri.v1.y), screenTri.v2.y);
//
//    screenTri.bottomRight.x = max(max(screenTri.v0.x, screenTri.v1.x), screenTri.v2.x);
//    screenTri.bottomRight.y = max(max(screenTri.v0.y, screenTri.v1.y), screenTri.v2.y);
//
//	return screenTri;
//}

__global__ void drawTriangle(float *redColorBuffer_d, float *greenColorBuffer_d,
		float *blueColorBuffer_d, float *depthBuffer_d)
{
	Triangle tri;
	tri.v0 = glm::vec3(0.0, 0.5, 0.0);
	tri.v1 = glm::vec3(-0.5, -0.5, 0.0);
	tri.v2 = glm::vec3(0.5, -0.25, 0.0);

	ScreenTriangle screenTri;

	screenTri.v0.x = ((tri.v0.x + 1) * (MY_WINDOW_SIZE - 1)) / 2;
	screenTri.v0.y = ((tri.v0.y + 1) * (MY_WINDOW_SIZE - 1)) / 2;
	screenTri.v0z = tri.v0.z;

	screenTri.v1.x = ((tri.v1.x + 1) * (MY_WINDOW_SIZE - 1)) / 2;
	screenTri.v1.y = ((tri.v1.y + 1) * (MY_WINDOW_SIZE - 1)) / 2;
    screenTri.v1z = tri.v1.z;

	screenTri.v2.x = ((tri.v2.x + 1) * (MY_WINDOW_SIZE - 1)) / 2;
	screenTri.v2.y = ((tri.v2.y + 1) * (MY_WINDOW_SIZE - 1)) / 2;
    screenTri.v2z = tri.v2.z;

    screenTri.normal = glm::normalize(glm::cross(glm::normalize(tri.v1 - tri.v0), glm::normalize(tri.v2 - tri.v0)));

	screenTri.topLeft.x = min(min(screenTri.v0.x, screenTri.v1.x), screenTri.v2.x);
    screenTri.topLeft.y = min(min(screenTri.v0.y, screenTri.v1.y), screenTri.v2.y);

    screenTri.bottomRight.x = max(max(screenTri.v0.x, screenTri.v1.x), screenTri.v2.x);
    screenTri.bottomRight.y = max(max(screenTri.v0.y, screenTri.v1.y), screenTri.v2.y);

	glm::vec3 light = glm::normalize(glm::vec3(1.0, 0.0, 1.0));

    float area = glm::determinant(glm::mat2(screenTri.v2 - screenTri.v0, screenTri.v1 - screenTri.v0));
    float color = max(glm::dot(screenTri.normal, light), 0.0f);
    
    for (int i = 0; i < MY_WINDOW_SIZE * MY_WINDOW_SIZE; i++)
    {
    	redColorBuffer_d[i] = 1.0;
    }

    for (int y = screenTri.topLeft.y; y < screenTri.bottomRight.y; ++y)
    {
        for (int x = screenTri.topLeft.x; x < screenTri.bottomRight.x; ++x)
        {
            if (x >= MY_WINDOW_SIZE || y >= MY_WINDOW_SIZE)
            {
                continue;
            }

            glm::ivec2 point(x, y);

            float a = glm::determinant(glm::mat2(screenTri.v2 - point, screenTri.v1 - point)) / area;
            float b = glm::determinant(glm::mat2(screenTri.v1 - point, screenTri.v0 - point)) / area;
            float g = glm::determinant(glm::mat2(screenTri.v0 - point, screenTri.v2 - point)) / area;

            if (a < 0 || b  < 0 || g < 0)
            {
                continue;
            }

            float depth = a * screenTri.v0z + b * screenTri.v1z + g * screenTri.v2z;

//            if (depth > depthBuffer_d[(y * MY_WINDOW_SIZE) + x])
            {
                redColorBuffer_d[(y * MY_WINDOW_SIZE) + x] = 0;
                greenColorBuffer_d[(y * MY_WINDOW_SIZE) + x] = 0;
                blueColorBuffer_d[(y * MY_WINDOW_SIZE) + x] = 1;

                depthBuffer_d[(y * MY_WINDOW_SIZE) + x] = depth;
            }
        }
    }
}

void writeImageToFile()
{
	Image img(MY_WINDOW_SIZE, MY_WINDOW_SIZE);

	for (int y = 0; y < MY_WINDOW_SIZE; ++y)
	{
		for (int x = 0; x < MY_WINDOW_SIZE; ++x)
		{
			color_t col;
			col.r = redColorBuffer[(y * MY_WINDOW_SIZE) + x];
			col.g = greenColorBuffer[(y * MY_WINDOW_SIZE) + x];
			col.b = blueColorBuffer[(y * MY_WINDOW_SIZE) + x];
			col.f = 1.0f;


			img.pixel(x, y, col);
		}
	}

	img.WriteTga("IMG.tga", false);
}

int main()
{
	float *redColorBuffer_d, *greenColorBuffer_d, *blueColorBuffer_d;
	float *depthBuffer_d;

  	hipMalloc((void **) &redColorBuffer_d, sizeof(float) * MY_WINDOW_SIZE * MY_WINDOW_SIZE);
  	hipMalloc((void **) &greenColorBuffer_d, sizeof(float) * MY_WINDOW_SIZE * MY_WINDOW_SIZE);
  	hipMalloc((void **) &blueColorBuffer_d, sizeof(float) * MY_WINDOW_SIZE * MY_WINDOW_SIZE);
  	hipMalloc((void **) &depthBuffer_d, sizeof(float) * MY_WINDOW_SIZE * MY_WINDOW_SIZE);

    memset(redColorBuffer, 0.0, sizeof(float) * MY_WINDOW_SIZE * MY_WINDOW_SIZE);
	hipMemcpy(redColorBuffer_d, redColorBuffer,
	 sizeof(float) * MY_WINDOW_SIZE * MY_WINDOW_SIZE, hipMemcpyHostToDevice);

    memset(greenColorBuffer, 1.0f, sizeof(float) * MY_WINDOW_SIZE * MY_WINDOW_SIZE);
    hipMemcpy(greenColorBuffer_d, greenColorBuffer,
     sizeof(float) * MY_WINDOW_SIZE * MY_WINDOW_SIZE, hipMemcpyHostToDevice);

    memset(blueColorBuffer, 0.0, sizeof(float) * MY_WINDOW_SIZE * MY_WINDOW_SIZE);
	hipMemcpy(blueColorBuffer_d, blueColorBuffer,
	 sizeof(float) * MY_WINDOW_SIZE * MY_WINDOW_SIZE, hipMemcpyHostToDevice);

    memset(depthBuffer, -1000.0, sizeof(float) * MY_WINDOW_SIZE * MY_WINDOW_SIZE);
	hipMemcpy(depthBuffer_d, depthBuffer,
     sizeof(float) * MY_WINDOW_SIZE * MY_WINDOW_SIZE, hipMemcpyHostToDevice);


//    for (int i = 0; i < MY_WINDOW_SIZE * MY_WINDOW_SIZE; ++i)
//    {
//        redColorBuffer[i] = 0.0;
//        greenColorBuffer[i] = 1.0;
//        blueColorBuffer[i] = 0.0;
//        depthBuffer[i] = -10000.0;
////      depthBuffer[i] = FLT_MIN;
//    }

    Mesh mesh = loadMesh("bunny10k.m");

//    for (int i = 0; i < mesh.tris.size(); ++i)
    {
        //std::cout << "Triangle: " << i << std::endl;
        drawTriangle<<<1, 1>>>(redColorBuffer_d, greenColorBuffer_d, blueColorBuffer_d, depthBuffer_d);
    }

	hipMemcpy(redColorBuffer, redColorBuffer_d,
	 sizeof(float) * MY_WINDOW_SIZE * MY_WINDOW_SIZE, hipMemcpyDeviceToHost);

    hipMemcpy(greenColorBuffer, greenColorBuffer_d,
     sizeof(float) * MY_WINDOW_SIZE * MY_WINDOW_SIZE, hipMemcpyDeviceToHost);

	hipMemcpy(blueColorBuffer, blueColorBuffer_d,
	 sizeof(float) * MY_WINDOW_SIZE * MY_WINDOW_SIZE, hipMemcpyDeviceToHost);

	hipMemcpy(depthBuffer, depthBuffer_d,
     sizeof(float) * MY_WINDOW_SIZE * MY_WINDOW_SIZE, hipMemcpyDeviceToHost);

	writeImageToFile();
	
	hipFree(redColorBuffer_d);
	hipFree(greenColorBuffer_d);
	hipFree(blueColorBuffer_d);
	hipFree(depthBuffer_d);

	printf("Done!");

	return 0;
}
